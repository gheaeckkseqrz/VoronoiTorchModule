#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define THREAD_COUNT 1024

__global__ void init(float *input, float *map, int w, int h)
{
  unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < w * h)
    {
      int x = index % w;
      int y = index / w;

      if (input[(0 * (w * h)) + (y * w) + x] != 0 || input[(1 * (w * h)) + (y * w) + x] != 0 || input[(2 * (w * h)) + (y * w) + x] != 0)
      	{
      	  map[(0 * (w * h)) + (y * w) + x] = (float)x;
      	  map[(1 * (w * h)) + (y * w) + x] = (float)y;
	  map[(2 * (w * h)) + (y * w) + x] = (float)0;
      	}
      else
      	{
      	  map[(0 * (w * h)) + (y * w) + x] = (float)-1;
      	  map[(1 * (w * h)) + (y * w) + x] = (float)-1;
      	  map[(2 * (w * h)) + (y * w) + x] = (float)-1;
      	}
    }
}

__global__ void run(float *map, float *out, int w, int h, unsigned int stride)
{
  unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < w * h)
    {
      unsigned int x = index % w;
      unsigned int y = index / w;

      int n[9][2] = {
	{-1, -1},
	{-1,  0},
	{-1,  1},
	{ 0, -1},
	{ 0,  0},
	{ 0,  1},
	{ 1, -1},
	{ 1,  0},
	{ 1,  1}
      };

      float bestScore = -1;
      int best = -1;
      for (unsigned int i=0 ; i < 9 ; ++i)
	{
	  int sx = x + (n[i][0] * stride);
	  int sy = y + (n[i][1] * stride);
	  if (sx >= 0 && sx < w && sy >= 0 && sy < h)
	    {
	      if (map[(2 * (w * h)) + (sy * w) + sx] >= 0)
		{
		  float score = sqrt(pow(map[(0 * (w * h)) + (sy * w) + sx] - x, 2) + pow(map[(1 * (w * h)) + (sy * w) + sx] - y, 2));
		  if (score < bestScore || bestScore < 0)
		    {
		      best = i;
		      bestScore = score;
		    }
		}
	    }
	}
      if (best >= 0)
	{
	  int sx = x + n[best][0] * stride;
	  int sy = y + n[best][1] * stride;
	  out[(0 * (w * h)) + (y * w) + x] = map[(0 * (w * h)) + (sy * w) + sx];
	  out[(1 * (w * h)) + (y * w) + x] = map[(1 * (w * h)) + (sy * w) + sx];
	  out[(2 * (w * h)) + (y * w) + x] = bestScore;
	}
    }
}

__global__ void finish(float *input, float *map, int w, int h)
{
  unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < w * h)
    {
      unsigned int x = index % w;
      unsigned int y = index / w;
      int mx = map[(0 * (w * h)) + (y * w) + x];
      int my = map[(1 * (w * h)) + (y * w) + x];

      if (mx >= 0 && my >= 0)
	{
	  input[(0 * (w * h)) + (y * w) + x] = input[(0 * (w * h)) + (my * w) + mx];
	  input[(1 * (w * h)) + (y * w) + x] = input[(1 * (w * h)) + (my * w) + mx];
	  input[(2 * (w * h)) + (y * w) + x] = input[(2 * (w * h)) + (my * w) + mx];
	}
    }
}

extern "C" void computeVoronoi(float *input, int w, int h)
{
  float *ping = NULL;
  float *pong = NULL;

  hipMalloc(&ping, 3 * w * h * sizeof(float)); // 3 Channels - Closest point X / Closest Point Y / Distance
  hipMalloc(&pong, 3 * w * h * sizeof(float)); // 3 Channels - Closest point X / Closest Point Y / Distance

  init<<<(w*h) / THREAD_COUNT + 1, THREAD_COUNT>>>(input, ping, w, h);
  init<<<(w*h) / THREAD_COUNT + 1, THREAD_COUNT>>>(input, pong, w, h);

  run<<<(w*h) / THREAD_COUNT + 1, THREAD_COUNT>>>(ping, pong, w, h, 128);
  run<<<(w*h) / THREAD_COUNT + 1, THREAD_COUNT>>>(pong, ping, w, h, 64);
  run<<<(w*h) / THREAD_COUNT + 1, THREAD_COUNT>>>(ping, pong, w, h, 32);
  run<<<(w*h) / THREAD_COUNT + 1, THREAD_COUNT>>>(pong, ping, w, h, 16);
  run<<<(w*h) / THREAD_COUNT + 1, THREAD_COUNT>>>(ping, pong, w, h, 8);
  run<<<(w*h) / THREAD_COUNT + 1, THREAD_COUNT>>>(pong, ping, w, h, 4);
  run<<<(w*h) / THREAD_COUNT + 1, THREAD_COUNT>>>(ping, pong, w, h, 2);
  run<<<(w*h) / THREAD_COUNT + 1, THREAD_COUNT>>>(pong, ping, w, h, 1);
  run<<<(w*h) / THREAD_COUNT + 1, THREAD_COUNT>>>(ping, pong, w, h, 1);

  finish<<<(w*h) / THREAD_COUNT + 1, THREAD_COUNT>>>(input, pong, w, h);

  hipFree(ping);
  hipFree(pong);
}

